
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <unistd.h>

#define FLOAT_SIZE 4


int main(int argc, char **argv) {
   int fp;
   int numElements;

   if (argc < 2) {
      fprintf(stderr, "Usage: %s <infile>\n", *argv);
      exit(1);
   }

   if ((fp = open(argv[1], O_RDONLY)) == -1) {
      perror("open");
      exit(1);
   }
   
   read(fp, &numElements, FLOAT_SIZE);

   

   return 0; 
   
}
