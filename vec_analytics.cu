#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <cmath>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <unistd.h>

#define FLOAT_SIZE 4
#define MAX_ELEMENTS 500000000
#define SPLIT_NUM 2

using std::cout;
using namespace thrust;

/* This macro was taken from the book CUDA by example. This code is used for
 * error checking */
static void HandleError(hipError_t err, const char *file, int line ) { 
   if (err != hipSuccess) {
      printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
      exit(1);
   }   
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

struct std_dev_help : public unary_function<float, float> {
   float standardMean;
   int *histoArr;
   std_dev_help(float mean, int *histo) {
      standardMean = mean;
      histoArr = histo;
   }
   __device__ float operator()(float &x) const {
      if (x > 99) {
         atomicAdd(&histoArr[99], 1);
      }
      else {
         atomicAdd(&histoArr[(int)x], 1);
      }
      return (x - standardMean) * (x - standardMean);
   }
};

void readElements(int fp, int numElements, float *elements, int size) {
   if (read(fp, elements, size) == -1) {
      perror("read");
      exit(1);
   }
}

int main(int argc, char **argv) {
   int fp;
   int readSize;
   int numElements;
   float mean;
   float standardDeviation;
   float min = 101;
   float max = -1;
   float newMin, newMax;
   int count = 0;

   if (argc < 3) {
      fprintf(stderr, "Usage: %s <infile> <histogram outfile>\n", *argv);
      exit(1);
   }

   if ((fp = open(argv[1], O_RDONLY)) == -1) {
      perror("open");
      exit(1);
   }

   if (read(fp, &numElements, FLOAT_SIZE) == -1) {
      perror("read");
      exit(1);
   }

   float *elements;
   
   if (numElements <= MAX_ELEMENTS) {
      readSize = numElements;
   }
   else {
      readSize = numElements / SPLIT_NUM;
   }
   printf("readSize: %d\n", readSize);
   elements = (float *)malloc(sizeof(float) * readSize);
   if (!elements) {
      perror("malloc");
      exit(1);
   }
  
   // Allocate space on device 
   device_vector<float> d_elements(readSize);
   int *dHistArr;
   HANDLE_ERROR(hipMalloc(&dHistArr, sizeof(int) * 100));
   HANDLE_ERROR(hipMemset(dHistArr, 0, sizeof(int) * 100));


   while (count < numElements) {
      // Read Elements in from file 
      readElements(fp, numElements, elements, readSize * FLOAT_SIZE);
      
      // Copy Elements to Device
      thrust::copy(elements, elements + readSize, d_elements.begin());
      // Compute Mean
      mean += reduce(d_elements.begin(), d_elements.end(), 0.0, 
       plus<float>());
      
      // Compute the Min and Max
      newMin = *min_element(d_elements.begin(), d_elements.end());
      newMax = *max_element(d_elements.begin(), d_elements.end());

      min = newMin < min ? newMin : min;
      max = newMax > max ? newMax : max;
      
      count += readSize;
   }

   lseek(fp, FLOAT_SIZE, SEEK_SET);
   count = 0;
   mean /= numElements;

   while (count < numElements) {

      if (readSize != numElements) {
         // Read Elements in from file 
         readElements(fp, numElements, elements, readSize * FLOAT_SIZE);
         // Copy Elements to Device
         thrust::copy(elements, elements + readSize, d_elements.begin());
      }

      // Standard Deviation and create histogram
      standardDeviation += transform_reduce(d_elements.begin(), 
       d_elements.end(), std_dev_help(mean, dHistArr), 0.0, plus<float>());
      count += readSize;
   }
   standardDeviation = sqrt(standardDeviation / numElements);
   
   int histoArr[100];
   HANDLE_ERROR(hipMemcpy(histoArr, dHistArr, sizeof(int) * 100, 
    hipMemcpyDeviceToHost));

   // Print Stats
   cout << "Count              : " << numElements << "\n";
   cout << "Minimum            : " << min << "\n";
   cout << "Maximum            : " << max << "\n";
   cout << "Mean               : " << mean << "\n"; 
   cout << "Standard Deviation : " << standardDeviation << "\n";

   // Write the Histogram File
   FILE *outfile = fopen(argv[2], "w");
   for (int x = 0; x < 100; x++) {
      fprintf(outfile, "%d, %d\n", x, histoArr[x]);
   }
   free(elements);
   return 0;
   
}
