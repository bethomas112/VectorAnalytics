#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <cmath>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <unistd.h>

#define FLOAT_SIZE 4

using namespace thrust;

struct std_dev_help : public unary_function<float, float> {
   float standardMean;
   int *histoArr;
   std_dev_help(float mean, int *histo) {
      standardMean = mean;
      histoArr = histo;
   }
   __device__ float operator()(float &x) const {
      atomicAdd(&histoArr[(int)x], 1);
      return (x - standardMean) * (x - standardMean);
   }
};

void readElements(int fp, int numElements, host_vector<float> *elements) {
   float temp;
   for (int i = 0; i < numElements; i++) {
      read(fp, &temp, FLOAT_SIZE);
      (*elements)[i] = temp;
   }
}

int main(int argc, char **argv) {
   int fp;
   int numElements;
   float mean;
   float standardDeviation;
   float min;
   float max;

   if (argc < 2) {
      fprintf(stderr, "Usage: %s <infile>\n", *argv);
      exit(1);
   }

   if ((fp = open(argv[1], O_RDONLY)) == -1) {
      perror("open");
      exit(1);
   }

   read(fp, &numElements, FLOAT_SIZE);
   host_vector<float> elements(numElements);
   
   readElements(fp, numElements, &elements);

   // Mean
   device_vector<float> d_elements = elements;
   mean = reduce(d_elements.begin(), d_elements.end(), 0.0, plus<float>()) / numElements;
   std::cout << mean << "\n";
   
   int *dHistArr;
   hipMalloc(&dHistArr, sizeof(int) * 100);
   hipMemset(dHistArr, 0, sizeof(int) * 100);
   // Standard Deviation and create histogram
   standardDeviation = sqrt(transform_reduce(d_elements.begin(), d_elements.end(), std_dev_help(mean, dHistArr), 0.0, plus<float>()) / numElements);
   std::cout << standardDeviation << "\n";

   min = *min_element(d_elements.begin(), d_elements.end());
   std::cout << min << "\n";

   max = *max_element(d_elements.begin(), d_elements.end());
   std::cout << max << "\n";
   
   int histoArr[100];
   hipMemcpy(histoArr, dHistArr, sizeof(int) * 100, hipMemcpyDeviceToHost);
   for (int x = 0; x < 100; x++) {
      std::cout << x << ", " << histoArr[x] << "\n";
   }
   return 0;
}
