#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>

#include "vec_analytics_dis.h"

using std::vector;
thrust::device_vector<float> d_elements(1);

/* This macro was taken from the book CUDA by example. This code is used for
 * error checking */
static void HandleError(hipError_t err, const char *file, int line ) { 
   if (err != hipSuccess) {
      printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
      exit(1);
   }   
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

struct std_dev_help : public thrust::unary_function<float, float> {
   float standardMean;
   int *histoArr;
   std_dev_help(float mean, int *histo) {
      standardMean = mean;
      histoArr = histo;
   }   
   __device__ float operator()(float &x) const {
      if (x > 99) {
         atomicAdd(&histoArr[99], 1); 
      }   
      else {
         atomicAdd(&histoArr[(int)x], 1); 
      }   
      return (x - standardMean) * (x - standardMean);
   }   
};


float computeMean(float *elementsNode, int numElements) {
   thrust::copy(elementsNode, numElements, d_elements.begin());
   return thrust::reduce(d_elements.begin(), d_elements.end(), 0.0, 
    thrust::plus<float>());
}

float computeStdDeviation(int *histo, float mean) {
   int *dHistArr; 
   float stdDeviation;
   
   HANDLE_ERROR(hipMalloc(&dHistArr, sizeof(int) * 100));
   HANDLE_ERROR(hipMemset(dHistArr, 0, sizeof(int) * 100));
   
   stdDeviation = thrust::transform_reduce(d_elements.begin(), 
    d_elements.end(), std_dev_help(mean, dHistArr), 0.0, 
    thrust::plus<float>());
  
   HANDLE_ERROR(hipMemcpy(histo, dHistArr, sizeof(int) * 100,
    hipMemcpyDeviceToHost));
   
   HANDLE_ERROR(hipFree(dHistArr));
   
   return stdDeviation;
}

float getMin(NULL) {
   return *min_element(d_elements.begin(), d_elements.end());
}

float getMax(NULL) {
   return *max_element(d_elements.begin(),d_elements.end());
}
